
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS_NUM 1
#define N ((int)sqrt(BLOCKS_NUM) * 960)
#define THREADS_NUM 4
#define BLOCK_SIZE (N/sqrt(BLOCKS_NUM))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct {
    int width;
    int height;
    int stride;
    int* elements;
} Matrix;

//inizializza le matrici
void initMatrix(Matrix A, Matrix B) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            A.elements[j + i*A.height] = 2;//((i + 1) * 10) + j + 1;
        }
    }
    for(int i=0; i< B.width; i++) {
        for(int j=0; j< B.width; j++) {
            B.elements[j + i*B.height] = 3;//((i + 1) * 10) + j + 1;
        }
    }
}

//inizializza le matrici
void printMatrix(Matrix A) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            printf("%d\t", A.elements[j + i*A.height]);
        }
        printf("\n");
    }
}


//Ritorna la sottomatrice Asub di dimensione BLOCK_SIZExBLOCK_SIZE
//di A che è localizzata col sottomatrici verso destra
// e row sottomatrici verso il basso
//dall'angolo in alto a sinistra di A
__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = blockDim.x; Asub.height = blockDim.x;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * blockDim.x * row
                                         + blockDim.x * col];
    return Asub;
}

//ritorna un elemento della matrice
__device__ int getElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}
//scrivi un elemento nella matrice
__device__ void setElement(Matrix A, int row, int col, int value)
{
    A.elements[row * A.stride + col] = value;
}

//Kernel per algoritmo di Cannon
__global__ void cannonKernel(Matrix A, Matrix B, Matrix C)
{   
    //riga e colonna del blocco
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    //Ogni blocco computa una sottomatrice Csub di C
    Matrix Csub = getSubMatrix(C, blockRow, blockCol);

    //Ogni thread computa una colonna di Csub
    //accumulando il risultato in Cvalue
    int Cvalue = 0;

    //riga del thread
    int col = threadIdx.x;

    //in base alla riga di A eseguo lo shift a sx
    //(col + n_shift) MOD (n_col);
    int a_shift = (blockCol + blockRow) % (gridDim.y);
    Matrix Asub = getSubMatrix(A, blockRow, a_shift);
    //in base alla colonna di B eseguo lo shift in su
    //(col + n_shift) MOD (n_col);
    int b_shift = (blockCol + blockRow) % (gridDim.x);
    Matrix Bsub = getSubMatrix(B, b_shift, blockCol);

    //Moltiplica ogni sottomatrice e accumula il risultato
    //Dopo ogni moltiplicazione esegui lo shift di uno
    //La moltiplicazione viene eseguito gridDim.x volte
    for (int i=0; i < gridDim.x; i++) {
        //moltiplica Asub e Bsub
        for (int k=0; k < blockDim.x; k++) {
            Cvalue = getElement(Csub, k, col);
            for(int j=0; j < blockDim.x; j++) {
                Cvalue += getElement(Asub, k, j) * getElement(Bsub, j, col);
            }
            setElement(Csub, k, col, Cvalue);
        }
        //in base alla riga di A eseguo lo shift a sx di 1
        //(col + n_shift) MOD (n_col);
        a_shift = (a_shift + 1)  % gridDim.x;
        Asub = getSubMatrix(A, blockRow, a_shift);
        //in base alla colonna di B eseguo lo shift in su di 1
        //(col + n_shift) MOD (n_col);
        b_shift = (b_shift + 1) % gridDim.x;
        Bsub = getSubMatrix(B, b_shift, blockCol);
    }
}

int main(void) {

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //matrici di partenza
    Matrix A, B, C;
    A.width = N; A.height = N; A.stride = N;
    B.width = N; B.height = N; B.stride = N;
    C.width = N; C.height = N; C.stride = N;
    //allocazione matrici
    size_t size = A.width * A.height * sizeof(int);
    A.elements = (int *)malloc(size);
    size = B.width * B.height * sizeof(int);
    B.elements = (int *)malloc(size);
    size = C.width * C.height * sizeof(int);
    C.elements = (int *)malloc(size);
    //inizializzazione matrici
    initMatrix(A, B);
    //matrici su GPU
    Matrix d_A, d_B, d_C;
    d_A.width = A.width; d_A.height = A.height; d_A.stride = N;
    d_B.width = B.width; d_B.height = B.height; d_B.stride = N;
    d_C.width = C.width; d_C.height = C.height; d_C.stride = N;
    //allocazione matrici su GPU
    size = d_A.width * d_A.height * sizeof(int);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    size = d_B.width * d_B.height * sizeof(int);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    size = d_C.width * d_C.height * sizeof(int);
    hipMalloc(&d_C.elements, size);

    //invoca kernel
    dim3 dimBlock(N/sqrt(BLOCKS_NUM));
    dim3 dimGrid(sqrt(BLOCKS_NUM), sqrt(BLOCKS_NUM));
    cannonKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost) );

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %3.1f ms/n\n\n", elapsedTime);
    
    //libera memoria
    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return 0;
}