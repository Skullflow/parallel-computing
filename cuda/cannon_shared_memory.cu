
#include <hip/hip_runtime.h>
#include <stdio.h>

#define n 30
#define N 1200

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct {
    int width;
    int height;
    int stride;
    int* elements;
} Matrix;

//inizializza le matrici
void initMatrix(Matrix A, Matrix B) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            A.elements[j + i*A.height] = (((i + 1) * 10) + j + 1) % 4 + 1;
        }
    }
    for(int i=0; i< B.width; i++) {
        for(int j=0; j< B.width; j++) {
            B.elements[j + i*B.height] = (((i + 1) * 10) + j + 1) % 6 + 2;
        }
    }
}

//inizializza le matrici
void printMatrix(Matrix A) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            printf("%d\t", A.elements[j + i*A.height]);
        }
        printf("\n");
    }
}

//Ritorna la sottomatrice Asub di dimensione BLOCK_SIZExBLOCK_SIZE
//di A che è localizzata col sottomatrici verso destra
// e row sottomatrici verso il basso
//dall'angolo in alto a sinistra di A
__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = n; Asub.height = n;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * n * row
                                         + n * col];
    return Asub;
}

//ritorna un elemento della matrice
__device__ int getElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}
//scrivi un elemento nella matrice
__device__ void setElement(Matrix A, int row, int col, int value)
{
    A.elements[row * A.stride + col] = value;
}

//Kernel per algoritmo di Cannon
__global__ void cannonKernel(Matrix A, Matrix B, Matrix C)
{   
    //riga e colonna del blocco
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    //Ogni blocco computa una sottomatrice Csub di C
    Matrix Csub = getSubMatrix(C, blockRow, blockCol);
    //Ogni thread computa una colonna di Csub
    //accumulando il risultato in Cvalue
    int Cvalue = 0;
    //riga del thread
    int col = threadIdx.x;
    int row =threadIdx.y;
    //in base alla riga di A eseguo lo shift a sx
    //(col + n_shift) MOD (n_col);
    int a_shift = (blockCol + blockRow) % (gridDim.y);
    Matrix Asub = getSubMatrix(A, blockRow, a_shift);
    //in base alla colonna di B eseguo lo shift in su
    //(col + n_shift) MOD (n_col);
    int b_shift = (blockCol + blockRow) % (gridDim.x);
    Matrix Bsub = getSubMatrix(B, b_shift, blockCol);
    //Utilizza memoria condivisa
    __shared__ int sA[n][n];
    __shared__ int sB[n][n];
    sA[row][col] = getElement(Asub, row, col);
    sB[row][col] = getElement(Bsub, row, col);
    __syncthreads();
    //Moltiplica ogni sottomatrice e accumula il risultato
    //Dopo ogni moltiplicazione esegui lo shift di uno
    //La moltiplicazione viene eseguito gridDim.x volte
    for (int i=0; i < gridDim.x; i++) {
        //moltiplica Asub e Bsub
        for(int j = 0; j < blockDim.x; j++) 
            Cvalue += sA[row][j] * sB[j][col];
        //in base alla riga di A eseguo lo shift a sx di 1
        //(col + n_shift) MOD (n_col);
        a_shift = (a_shift + 1)  % gridDim.x;
        Asub = getSubMatrix(A, blockRow, a_shift);
        //in base alla colonna di B eseguo lo shift in su di 1
        //(col + n_shift) MOD (n_col);
        b_shift = (b_shift + 1) % gridDim.x;
        Bsub = getSubMatrix(B, b_shift, blockCol);
        //setta memoria condivisa con i prossimi valori
        sA[row][col] = getElement(Asub, row, col);
        sB[row][col] = getElement(Bsub, row, col);
        __syncthreads();
    }
    setElement(Csub, row, col, Cvalue);
}

int main(void) {

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //matrici di partenza
    Matrix A, B, C;
    A.width = N; A.height = N; A.stride = N;
    B.width = N; B.height = N; B.stride = N;
    C.width = N; C.height = N; C.stride = N;
    //allocazione matrici
    size_t size = A.width * A.height * sizeof(int);
    A.elements = (int *)malloc(size);
    size = B.width * B.height * sizeof(int);
    B.elements = (int *)malloc(size);
    size = C.width * C.height * sizeof(int);
    C.elements = (int *)malloc(size);
    //inizializzazione matrici
    initMatrix(A, B);
    //matrici su GPU
    Matrix d_A, d_B, d_C;
    d_A.width = A.width; d_A.height = A.height; d_A.stride = N;
    d_B.width = B.width; d_B.height = B.height; d_B.stride = N;
    d_C.width = C.width; d_C.height = C.height; d_C.stride = N;
    //allocazione matrici su GPU
    size = d_A.width * d_A.height * sizeof(int);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    size = d_B.width * d_B.height * sizeof(int);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    size = d_C.width * d_C.height * sizeof(int);
    hipMalloc(&d_C.elements, size);
    //invoca kernel
    dim3 dimBlock(n,n);
    dim3 dimGrid(N/n, N/n);
    printf("N: %d\n", N);
    printf("GRIGLIA: [%d,%d]\n", N/n, N/n);
    printf("THREADS: [%d,%d]\n", n, n);
    cannonKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    gpuErrchk( hipPeekAtLastError() );
    //Copia C da device a host
    gpuErrchk( hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost) );
    //registra stop
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %3.1f ms/n\n\n", elapsedTime);
    //libera memoria
    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return 0;
}