
#include <hip/hip_runtime.h>
#include <stdio.h>

#define Q 5
#define BLOCKS_NUM (Q*Q*Q)
#define N (Q * 240)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct {
    int width;
    int height;
    int stride;
    int* elements;
} Matrix;

//inizializza le matrici
void initMatrix(Matrix A, Matrix B) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            A.elements[j + i*A.height] = 2;//((i + 1) * 10) + j + 1;
        }
    }
    for(int i=0; i< B.width; i++) {
        for(int j=0; j< B.width; j++) {
            B.elements[j + i*B.height] = 3;//((i + 1) * 10) + j + 1;
        }
    }
}

//inizializza le matrici
void printMatrix(Matrix A) {
    for(int i=0; i< A.width; i++) {
        for(int j=0; j< A.width; j++) {
            printf("%d\t", A.elements[j + i*A.height]);
        }
        printf("\n");
    }
}

//Ritorna la sottomatrice Asub di dimensione BLOCK_SIZExBLOCK_SIZE
//di A che è localizzata col sottomatrici verso destra
// e row sottomatrici verso il basso
//dall'angolo in alto a sinistra di A
__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = blockDim.x; Asub.height = blockDim.x;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * blockDim.x * row
                                         + blockDim.x * col];
    return Asub;
}

//ritorna un elemento della matrice
__device__ int getElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

__device__ void multiplyAndReduce(Matrix A, Matrix B, Matrix C)
{
    int col = threadIdx.x;
    for (int k=0; k < blockDim.x; k++) {
        int Cvalue = 0;
        for(int j=0; j < blockDim.x; j++) 
            Cvalue += getElement(A, k, j) * getElement(B, j, col);
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = k + blockIdx.y * blockDim.x;
        int offset = x + y * blockDim.x * gridDim.x;
        atomicAdd(&C.elements[offset], Cvalue);
    }
}

//Kernel per algoritmo DNS
__global__ void dnsKernel(Matrix A, Matrix B, Matrix C)
{
     //riga colonna e profondità del blocco
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int blockDeep = blockIdx.z;
    //Ricevi la sottomatrice di A in base all'asse z
    Matrix Asub = getSubMatrix(A, blockRow, blockDeep);
    //Ricevi la sottomatrice di B in base all'asse z
    Matrix Bsub = getSubMatrix(A, blockDeep, blockCol);
    //Moltiplica e riduci con una operazione di somma sull'asse z
    multiplyAndReduce(Asub, Bsub, C);
}

int main(void) {

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //matrici di partenza
    Matrix A, B, C;
    A.width = N; A.height = N; A.stride = N;
    B.width = N; B.height = N; B.stride = N;
    C.width = N; C.height = N; C.stride = N;
    //allocazione matrici
    size_t size = A.width * A.height * sizeof(int);
    A.elements = (int *)malloc(size);
    size = B.width * B.height * sizeof(int);
    B.elements = (int *)malloc(size);
    size = C.width * C.height * sizeof(int);
    C.elements = (int *)malloc(size);
    //inizializzazione matrici
    initMatrix(A, B);
    //matrici su GPU
    Matrix d_A, d_B, d_C;
    d_A.width = A.width; d_A.height = A.height; d_A.stride = N;
    d_B.width = B.width; d_B.height = B.height; d_B.stride = N;
    d_C.width = C.width; d_C.height = C.height; d_C.stride = N;
    //allocazione matrici su GPU
    size = d_A.width * d_A.height * sizeof(int);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    size = d_B.width * d_B.height * sizeof(int);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    size = d_C.width * d_C.height * sizeof(int);
    hipMalloc(&d_C.elements, size);
    //invoca kernel
    dim3 dimBlock(N/Q);
    dim3 dimGrid(Q, Q, Q);
    dnsKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost) );
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %3.1f ms/n\n\n", elapsedTime);
    //libera memoria
    free(A.elements);
    free(B.elements);
    free(C.elements);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return 0;
}